#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"

using std::cout;
using std::endl;

void helloCuda(void);

__global__ static void kernelHello(void);
__device__ static void doSomethingHello(void);

void helloCuda(void)
    {
    cout << endl << "[Hello Cuda]" << endl;

    // Organisation logique hierarchique des threads
    //
    // 		L'architecture logique des threads en cuda est hierarchique sur 3 niveaux:
    //
    //			(Niveau 1) 	D'une grille 2D de blocks.
    //					La dimension de cette grille G est [gridDim.x, gridDim.y]
    //					La dimemsion max d�pend du GPU . Example : [65535,65535,1]
    //					Un block est localis� par 2 index : (blockIdx.x,blockIdx.y)
    //					Cette grille poss�de une m�moire
    //						global
    //						constant
    //
    //			(Niveau 2) 	Chaque block est constitu� d'une matrice3D de threads
    //					La dimension de ce block est [blockDim.x, blockDim.y, blockDim.z]
    //					La dimemsion max d�pend du GPU . Example : [1024,1024,64]
    //					Un thread est localis� par 3 index : (threadIdx.x,threadIdx.y,threadIdx.y)
    //					Un block poss�de une m�moire
    //						shared (petite, rapide, accessible only by thread of block)
    //
    //			(Niveau 3)	Un thread est atomique, ie non decomposable
    //					Un thread poss�de une m�moire
    //						local
    // Terminologie:
    //
    //		blockIdx = blockIndex
    // 		threadIdx = threadIndex
    //
    //
    // Type:
    // 		Dim3 est une structure � 3 champs : (x,y,z)
    // 		Dim3 est utilis� pour repr�senter �
    //			(1) Des index
    //				blockId	(avec z=1 toujours)
    //				threadId
    //			(2)Des dimension
    //				gridDim (avec z=1 toujours)
    //				blockDim
    //

    dim3 blockPerGrid = dim3(1, 1); // ou dim3(1, 1) // mais pas  dim3(1, 1,0)
    dim3 threadPerBlock = dim3(1, 1, 1); // ou dim3(1,1)

    print(blockPerGrid, threadPerBlock);
    assertDimLegal(blockPerGrid, threadPerBlock);

    // 1 block de 1 Thread sur le GPU
kernelHello<<<blockPerGrid,threadPerBlock>>>(); // 1 block de 1 Thread sur le GPU
    // addScalar<<<1,1>>>(a,b,ptrDev_c); // syntaxe simplifi�
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void kernelHello(void)
    {
    doSomethingHello();
    }

/**
 * Can be call only by device
 * inliner by nvcc (nvidia compiler)
 */
__device__ void doSomethingHello(void)
    {
    // rien
    }
