#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaTools.h"

__global__ static void kernelHello(void);
__device__ static void doSomethingHello(void);

extern void launchMandelBrot(void){
    std::cout << "Launch MandelBrot (CUDA Version)" << std::endl << std::endl;

    dim3 blockPerGrid = dim3(1, 1); // ou dim3(1, 1) // mais pas  dim3(1, 1,0)
    dim3 threadPerBlock = dim3(1, 1, 1); // ou dim3(1,1)

    std::cout << "Dimensions" << std::endl;
    print(blockPerGrid, threadPerBlock);

    assertDimLegal(blockPerGrid, threadPerBlock);

    kernelHello<<<blockPerGrid,threadPerBlock>>>();
}

__global__ static void kernelHello(void){
    doSomethingHello();
}

__device__ void doSomethingHello(void){
    // rien
}
