#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include "cudaTools.h"

using std::cout;
using std::endl;

__global__ static void kernelHello(void);
__device__ static void doSomethingHello(void);

extern void launchMandelBrot(void){
    cout << endl << "[Hello Cuda]" << endl;

    dim3 blockPerGrid = dim3(1, 1); // ou dim3(1, 1) // mais pas  dim3(1, 1,0)
    dim3 threadPerBlock = dim3(1, 1, 1); // ou dim3(1,1)

    print(blockPerGrid, threadPerBlock);
    assertDimLegal(blockPerGrid, threadPerBlock);

    kernelHello<<<blockPerGrid,threadPerBlock>>>();
}

__global__ static void kernelHello(void)
{
    doSomethingHello();
}

__device__ void doSomethingHello(void)
    {
    // rien
    }
