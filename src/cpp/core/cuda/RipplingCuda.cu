#include "hip/hip_runtime.h"
#include <iostream>
#include "GLImageCudas.h"
#include "ColorTools_Device.h"

__global__ static void kernelAnimationHSB(uchar4* ptrDevPixels, int w, int h, float t);

__device__ static float color(int w, int h, float x, float y, float t);
__device__ static float d(int w, int h, float x, float y);

void useKernelAnimationHSB(uchar4* ptrDevPixels, int w, int h, float t){
    dim3 blockPerGrid = dim3(32, 32, 1);
    dim3 threadPerBlock = dim3(16, 16, 1);

    kernelAnimationHSB<<<blockPerGrid,threadPerBlock>>>(ptrDevPixels, w, h, t);
}

__global__ static void kernelAnimationHSB(uchar4* ptrDevPixels, int w, int h, float t){
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    int nbThreadY = gridDim.y * blockDim.y;
    int nbThreadX = gridDim.x * blockDim.x;
    int nbThreadCuda = nbThreadY * nbThreadX;

    int tid = j +  (i * nbThreadX);
    int pixelI;
    int pixelJ;

    while(tid < (w * h)){
	pixelI = tid / w;
	pixelJ = tid - w * pixelI;

	float c = color(w, h, pixelI, pixelJ, t);

	ptrDevPixels[tid].x = c;
	ptrDevPixels[tid].y = c;
	ptrDevPixels[tid].z = c;
	ptrDevPixels[tid].w = 255;

	tid += nbThreadCuda;
    }
}

__device__ static float color(int w, int h, float x, float y, float t){
    return 128 + 127 * ((cos(d(w, h, x,y) / (float)10 -(t / (float)7))) / (d(w, h, x, y) / 10 + 1));
}

__device__ static float d(int w, int h, float x, float y){
    float fx = x - (w / 2);
    float fy = y - (h / 2);

    return sqrt(fx * fx + fy * fy);
}
